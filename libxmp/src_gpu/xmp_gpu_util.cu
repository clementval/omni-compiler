#include "xmp_gpu_internal.h"

void _XMP_gpu_alloc(void **addr, size_t size) {
  if (hipMalloc(addr, size) != hipSuccess) {
    _XMP_fatal("failed to allocate data on GPU");
  }
}

void _XMP_gpu_free(void *addr) {
  if (hipFree(addr) != hipSuccess) {
    _XMP_fatal("failed to free data on GPU");
  }
}
