#include "acc_internal.h"
#include "acc_gpu_internal.h"
#include <stdio.h>
#include <stdbool.h>

void _ACC_gpu_alloc(void **addr, size_t size)
{
  //printf("_ACC_gpu_alloc\n");
  _ACC_DEBUG("alloc addr=%p, size=%zd\n", addr, size)
  _ACC_gpu_init_current_device_if_not_inited();
  hipError_t cuda_err = hipMalloc(addr, size);
  if (cuda_err != hipSuccess) {
    printf("failed to allocate data on GPU\n");
    _ACC_gpu_fatal(cuda_err);
    //_ACC_fatal("failed to allocate data on GPU");
  }
}

void _ACC_gpu_free(void *addr)
{
  //printf("_ACC_gpu_free\n");
  hipError_t cuda_err = hipFree(addr);
  if (cuda_err != hipSuccess) {
    printf("failed to free data on GPU(%d)\n",(int)cuda_err);
    _ACC_gpu_fatal(cuda_err);
    //_ACC_fatal("failed to free data on GPU");
  }
}

void _ACC_gpu_malloc(void **addr, size_t size)
{
  _ACC_gpu_alloc(addr, size);
}

void _ACC_gpu_calloc(void **addr, size_t size)
{
  //printf("_ACC_gpu_calloc()\n");
  _ACC_gpu_alloc(addr, size);

  hipError_t cuda_err = hipMemset(*addr, 0, size);
  if(cuda_err != hipSuccess){
    _ACC_fatal("failed to clear data on GPU");
  }
}

void _ACC_gpu_copy(void *host_addr, void *device_addr, size_t size, int direction){
  hipError_t cuda_err;
  if(direction == _ACC_GPU_COPY_HOST_TO_DEVICE){
	_ACC_DEBUG("copy host(%p) to dev(%p), size(%zd)\n", host_addr, device_addr, size)
    cuda_err = hipMemcpy(device_addr, host_addr, size, hipMemcpyHostToDevice);
  }else if(direction == _ACC_GPU_COPY_DEVICE_TO_HOST){
	_ACC_DEBUG("copy dev(%p) to host(%p), size(%zd)\n", device_addr, host_addr, size)
    cuda_err = hipMemcpy(host_addr, device_addr, size, hipMemcpyDeviceToHost);
  }else{
    _ACC_fatal("invaild direction in 'gpu_copy'");
  }
  
  if(cuda_err != hipSuccess){
    const char *err_str = hipGetErrorString(cuda_err);
    _ACC_fatal( (char *)err_str );
  }
}

void _ACC_gpu_copy_async(void *host_addr, void *device_addr, size_t size, int direction, int id){
  //printf("_ACC_gpu_copy_async\n");
  hipError_t cuda_err;
  hipStream_t stream = _ACC_gpu_get_stream(id);

  switch(direction){
  case _ACC_GPU_COPY_HOST_TO_DEVICE:
    cuda_err = hipMemcpyAsync(device_addr, host_addr, size, hipMemcpyHostToDevice, stream);
    break;
  case _ACC_GPU_COPY_DEVICE_TO_HOST:
    cuda_err = hipMemcpyAsync(host_addr, device_addr, size, hipMemcpyDeviceToHost, stream);
    break;
  default:
    _ACC_fatal("invaild direction in 'gpu_copy_async'");
  }
  
  if(cuda_err != hipSuccess){
    _ACC_gpu_fatal(cuda_err);
  }
}

void _ACC_gpu_register_memory(void *host_addr, size_t size){
  //printf("register_memory\n");
  hipError_t cuda_err = hipHostRegister(host_addr, size, hipHostRegisterPortable);
  if( cuda_err != hipSuccess){
    _ACC_gpu_fatal(cuda_err);
    //return false;
  }else{
    //return true;
  }
}

void _ACC_gpu_unregister_memory(void *host_addr){
  //printf("unregister_memory\n");
  hipError_t cuda_err = hipHostUnregister(host_addr);
  if( cuda_err != hipSuccess){
    _ACC_gpu_fatal(cuda_err);
    //return false;
  }else{
    //return true;
  }
}


void _ACC_gpu_fatal(hipError_t error)
{
  _ACC_fatal(hipGetErrorString(error));
}

/*
int _ACC_gpu_get_num_devices()
{
  int count;
  hipError_t error = hipGetDeviceCount(&count);
  if(error != hipSuccess){
    _ACC_gpu_fatal(error);
  }
  return count;
}
*/

bool _ACC_gpu_is_pagelocked(void *p)
{
  unsigned int flags;
  hipHostGetFlags(&flags, p);
  hipError_t error = hipGetLastError();
  return (error == hipSuccess);
}

void *_ACC_alloc_pinned(size_t size){
  void *addr;
  hipError_t err = hipHostMalloc((void**)&addr, size);
  if(err != hipSuccess){
    _ACC_gpu_fatal(err);
  }
  return addr;
}

void _ACC_free_pinned(void *p)
{
  hipError_t err = hipHostFree(p);
  if(err != hipSuccess){
    _ACC_gpu_fatal(err);
  }
}
